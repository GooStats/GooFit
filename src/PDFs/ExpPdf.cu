#include "goofit/PDFs/ExpPdf.hh"

EXEC_TARGET fptype device_Exp (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x = evt[indices[2 + indices[0]]]; 
  fptype alpha = p[indices[1]];

  fptype ret = EXP(alpha*x); 
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_Exp = device_Exp; 

__host__ ExpPdf::ExpPdf (std::string n, Variable* _x, Variable* alpha, Variable* offset) 
  : GooPdf(_x, n) 
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(alpha));
  GET_FUNCTION_ADDR(ptr_to_Exp);
  initialise(pindices); 
}


__host__ fptype ExpPdf::integrate (fptype lo, fptype hi) const {
  fptype alpha = host_params[host_indices[parameters + 1]]; 

  if (0 == alpha) {
    // This gives a constant 1 all across the range
    return (hi - lo); 
  }

  fptype ret = EXP(alpha*hi) - EXP(alpha*lo);
  ret /= alpha; 
  return ret; 
}

