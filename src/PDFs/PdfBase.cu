#include "hip/hip_runtime.h"
#include "goofit/BinnedDataSet.h"
#include "goofit/FitControl.h"
#include "goofit/PdfBase.h"
#include "goofit/UnbinnedDataSet.h"
#include "goofit/Variable.h"

// This is code that belongs to the PdfBase class, that is,
// it is common across all implementations. But it calls on device-side
// functions, and due to the nvcc translation-unit limitations, it cannot
// sit in its own object file; it must go in the CUDAglob.cu. So it's
// off on its own in this inline-cuda file, which GooPdf.cu
// should include.

__host__ void PdfBase::copyParams(const std::vector<double> &pars) {
  // copyParams method performs eponymous action!

  for (unsigned int i = 0; i < pars.size(); ++i) {
    host_params[i] = pars[i];

    if (std::isnan(host_params[i])) {
      std::cout << " agh, parameter is NaN, die " << i << std::endl;
      abortWithCudaPrintFlush(__FILE__, __LINE__, "NaN in parameter");
    }
  }

  MEMCPY_TO_SYMBOL(cuda_array, host_params, pars.size() * sizeof(fptype), 0, hipMemcpyHostToDevice);
}

__host__ void PdfBase::copyParams() const {
  // Copies values of Variable objects
  parCont pars;
  getParameters(pars);
  int maxIndex = -1;
  for (auto &par: pars) {
    if (maxIndex < par->getIndex()) {
      maxIndex = par->getIndex();
    }
  }
  std::vector<double> values(host_params, host_params + maxIndex + 1);
  for (auto &par: pars) {
    values[par->getIndex()] = par->value;
  }
  copyParams(values);
}

__host__ void PdfBase::copyNormFactors() {
  //MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice);
  SYNCH();// Ensure normalisation integrals are finished
}

__host__ void PdfBase::initialiseIndices(std::vector<unsigned int> pindices) {
  // Structure of the individual index array: Number of parameters, then the indices
  // requested by the subclass (which will be interpreted by the subclass kernel),
  // then the number of observables, then the observable indices. Notice that the
  // observable indices are not set until 'setIndices' is called, usually from setData;
  // here we only reserve space for them by setting totalParams.
  // This is to allow index sharing between PDFs - all the PDFs must be constructed
  // before we know what observables exist.

  if (totalParams + pindices.size() >= maxIndicies) {
    std::cout << "Major problem with pindices size: " << totalParams << " + " << pindices.size() << " >= " << maxIndicies << std::endl;
  }

  assert(totalParams + pindices.size() < maxIndicies);
  host_indices[totalParams] = pindices.size();
  for (unsigned int i = 1; i <= host_indices[totalParams]; ++i) {
    host_indices[totalParams + i] = pindices[i - 1];
  }
  host_indices[totalParams + pindices.size() + 1] = observables.size();

  parameters = totalParams;
  totalParams += (2 + pindices.size() + observables.size());
  pdfName[parameters] = getName();

  /*
  std::cout << " | "
	    << parameters << " "
	    << totalParams << " "
	    << cuda_array << " "
	    << paramIndices << " "
	    << std::endl;
  */
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams * sizeof(unsigned int), 0, hipMemcpyHostToDevice);
}

__host__ void PdfBase::setData(std::vector<std::map<Variable *, fptype>> &__attribute__((__unused__)) data) {
  abortWithCudaPrintFlush(__FILE__, __LINE__, "not possible to use this method. one Variable could correspond to different index, to save space");
}

__host__ void PdfBase::recursiveSetIndices() {
  for (auto &component: components) {
    component->recursiveSetIndices();
  }

  int numParams = host_indices[parameters];
  int counter = 0;
  for (auto v = obsBegin(); v != obsEnd(); ++v) {
    host_indices[parameters + 2 + numParams + counter] = counter;
    counter++;
  }
  generateNormRange();
}

__host__ void PdfBase::setIndices() {
  recursiveSetIndices();
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams * sizeof(unsigned int), 0, hipMemcpyHostToDevice);
}

__host__ void PdfBase::setData(UnbinnedDataSet *data) {
  if (pdfId == -1) {
    registerPdf();
  }
  if (dev_event_array[pdfId] != nullptr) {
    gooFree(dev_event_array[pdfId]);
    SYNCH();
    dev_event_array[pdfId] = nullptr;
  }

  setIndices();
  int dimensions = observables.size();
  numEntries = data->getNumEvents();
  numEvents = numEntries;
  if (fitControl->binnedFit()) {
    setFitControl(new UnbinnedNllFit());
  }

  if (numEntries > 0) {
    auto *host_array = new fptype[numEntries * dimensions];
    for (unsigned int i = 0; i < numEntries; ++i) {
      int j = 0;
      for (auto v = obsBegin(); v != obsEnd(); ++v) {
        fptype currVal = data->getValue((*v), i);
        int position = host_indices[parameters + 2 + host_indices[parameters] + j];
        host_array[i * dimensions + position] = currVal;
        ++j;
      }
    }

    gooMalloc(reinterpret_cast<void **>(&(dev_event_array[pdfId])), dimensions * numEntries * sizeof(fptype));
    MEMCPY(dev_event_array[pdfId], host_array, dimensions * numEntries * sizeof(fptype), hipMemcpyHostToDevice);
    SYNCH();
    MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice);

    delete[] host_array;
  }
}

__host__ void PdfBase::setData(BinnedDataSet *data) {
  if (pdfId == -1) {
    registerPdf();
  }
  if (dev_event_array[pdfId] != nullptr) {
    gooFree(dev_event_array[pdfId]);
    dev_event_array[pdfId] = nullptr;
  }

  setIndices();
  numEvents = 0;
  numEntries = data->getNumBins();
  if (numEntries <= 0) {
    abortWithCudaPrintFlush(__FILE__, __LINE__, "0 entries. check the numbins of the variable of your data set", this);
  }
  int dimensions = 2 + observables.size();// Bin center (x,y, ...), bin value, and bin volume.
  if (!fitControl->binnedFit()) {
    setFitControl(new BinnedNllFit());
  }

  if (numEntries > 0) {
    auto *host_array = new fptype[numEntries * dimensions];

    for (unsigned int i = 0; i < numEntries; ++i) {
      int j = 0;
      for (auto v = obsBegin(); v != obsEnd(); ++v) {
        int position = host_indices[parameters + 2 + host_indices[parameters] + j];
        assert(position == j);
        host_array[i * dimensions + position] = data->getBinCenter((*v), i);
        ++j;
      }

      host_array[i * dimensions + observables.size() + 0] = data->getBinContent(i);
      host_array[i * dimensions + observables.size() + 1] = fitControl->binErrors() ? data->getBinError(i) : data->getBinVolume(i);
      numEvents += data->getBinContent(i);
    }

    gooMalloc(reinterpret_cast<void **>(&(dev_event_array[pdfId])), dimensions * numEntries * sizeof(fptype));
    MEMCPY(dev_event_array[pdfId], host_array, dimensions * numEntries * sizeof(fptype), hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice);
    delete[] host_array;
  }
}

__host__ void PdfBase::generateNormRange() {
  if (normRanges != nullptr) {
    gooFree(normRanges);
  }
  gooMalloc(reinterpret_cast<void **>(&normRanges), 3 * observables.size() * sizeof(fptype));

  auto *host_norms = new fptype[3 * observables.size()];
  int counter = 0;// Don't use index in this case to allow for, eg,
  // a single observable whose index is 1; or two observables with indices
  // 0 and 2. Make one array per functor, as opposed to variable, to make
  // it easy to pass MetricTaker a range without worrying about which parts
  // to use.
  for (auto v = obsBegin(); v != obsEnd(); ++v) {
    host_norms[3 * counter + 0] = (*v)->lowerlimit;
    host_norms[3 * counter + 1] = (*v)->upperlimit;
    host_norms[3 * counter + 2] = integrationBins > 0 ? integrationBins : (*v)->numbins;
    counter++;
  }

  MEMCPY(normRanges, host_norms, 3 * observables.size() * sizeof(fptype), hipMemcpyHostToDevice);
  delete[] host_norms;
}

void PdfBase::clearCurrentFit() const {
  totalParams = 0;
  gooFree(dev_event_array[pdfId]);
  dev_event_array[pdfId] = nullptr;
}

__host__ void PdfBase::printProfileInfo(bool __attribute__((__unused__)) topLevel) {
#ifdef PROFILING
  if (topLevel) {
    hipError_t err = MEMCPY_FROM_SYMBOL(host_timeHist, timeHistogram, 10000 * sizeof(fptype), 0, hipMemcpyDeviceToHost);
    if (hipSuccess != err) {
      std::cout << "Error on copying timeHistogram: " << hipGetErrorString(err) << std::endl;
      return;
    }

    std::cout << getName() << " : " << getFunctionIndex() << " " << host_timeHist[100 * getFunctionIndex() + getParameterIndex()] << std::endl;
    for (unsigned int i = 0; i < components.size(); ++i) {
      components[i]->printProfileInfo(false);
    }
  }
#endif
}


gooError gooMalloc(void **target, size_t bytes) {
  // Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_SYSTEM != THRUST_DEVICE_SYSTEM_CUDA
  target[0] = malloc(bytes);
  if (target[0] != nullptr) {
    return gooSuccess;
  }
  return gooErrorMemoryAllocation;
#else
  if (hipMalloc(target, bytes) != hipSuccess)
    throw std::runtime_error("cannot allocate enough memory on the device");
  return (gooError) hipSuccess;
#endif
}

gooError gooFree(void *ptr) {
  gooError ret;
  // Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_SYSTEM != THRUST_DEVICE_SYSTEM_CUDA
  free(ptr);
  ret = gooSuccess;
#else
  ret = (gooError) hipFree(ptr);
#endif
  ptr = nullptr;
  return ret;
}
